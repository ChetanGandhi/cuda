
#include <hip/hip_runtime.h>
// #include <stdio.h>

#define BLOCKS_COUNT (2048 * 2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *numberOne, int *numberTwo, int *addition, int count)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < count)
    {
        addition[index] = numberOne[index] + numberTwo[index];
    }
}

void random_ints(int* dest, int count)
{
    int counter = 0;
    for (counter = 0; counter < count; ++counter)
    {
        dest[counter] = rand();
    }
}

int main(void)
{
    // The host variables.
    int *host_numberOne, *host_numberTwo, *host_addition;

    // The device variables.
    int *device_numberOne, *device_numberTwo, *device_addition;

    // Size of variable per block
    int size = BLOCKS_COUNT * sizeof(int);

    // int counter = 0;

    // Allocate memory on device for device variables.
    hipMalloc((void **)&device_numberOne, size);
    hipMalloc((void **)&device_numberTwo, size);
    hipMalloc((void **)&device_addition, size);

    // Initialize host variables;
    host_numberOne = (int *)malloc(size);
    random_ints(host_numberOne, BLOCKS_COUNT);

    host_numberTwo = (int *)malloc(size);
    random_ints(host_numberTwo, BLOCKS_COUNT);

    host_addition = (int *)malloc(size);

    // Copy host variables to device memory.
    hipMemcpy(device_numberOne, host_numberOne, size, hipMemcpyHostToDevice);
    hipMemcpy(device_numberTwo, host_numberTwo, size, hipMemcpyHostToDevice);

    // Invoke add kernel.
    add<<<(BLOCKS_COUNT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(device_numberOne, device_numberTwo, device_addition, BLOCKS_COUNT);

    // Copy device variable to host memory.
    hipMemcpy(host_addition, device_addition, size, hipMemcpyDeviceToHost);

    // for(counter = 0; counter < BLOCKS_COUNT; ++counter)
    // {
    //     printf("%5d + %5d = %5d\n", host_numberOne[counter], host_numberTwo[counter], host_addition[counter]);
    // }

    // Clean up, free all device allocated memory.
    free(host_numberOne);
    free(host_numberTwo);
    free(host_addition);
    hipFree(device_numberOne);
    hipFree(device_numberTwo);
    hipFree(device_addition);

    return 0;
}
