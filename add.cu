
#include <hip/hip_runtime.h>
// #include <stdio.h>

#define BLOCKS_COUNT 512

__global__ void add(int *numberOne, int *numberTwo, int *addition)
{
    addition[blockIdx.x] = numberOne[blockIdx.x] + numberTwo[blockIdx.x];
}

void random_ints(int* dest, int count)
{
    int counter = 0;
    for (counter = 0; counter < count; ++counter)
    {
        dest[counter] = rand();
    }
}

int main(void)
{
    // The host variables.
    int *host_numberOne, *host_numberTwo, *host_addition;

    // The device variables.
    int *device_numberOne, *device_numberTwo, *device_addition;

    // Size of variable per block
    int size = BLOCKS_COUNT * sizeof(int);

    // int counter = 0;

    // Allocate memory on device for device variables.
    hipMalloc((void **)&device_numberOne, size);
    hipMalloc((void **)&device_numberTwo, size);
    hipMalloc((void **)&device_addition, size);

    // Initialize host variables;
    host_numberOne = (int *)malloc(size);
    random_ints(host_numberOne, BLOCKS_COUNT);

    host_numberTwo = (int *)malloc(size);
    random_ints(host_numberTwo, BLOCKS_COUNT);

    host_addition = (int *)malloc(size);

    // Copy host variables to device memory.
    hipMemcpy(device_numberOne, host_numberOne, size, hipMemcpyHostToDevice);
    hipMemcpy(device_numberTwo, host_numberTwo, size, hipMemcpyHostToDevice);

    // Invoke add kernel.
    add<<<BLOCKS_COUNT, 1>>>(device_numberOne, device_numberTwo, device_addition);

    // Copy device variable to host memory.
    hipMemcpy(host_addition, device_addition, size, hipMemcpyDeviceToHost);

    // for(counter = 0; counter < BLOCKS_COUNT; ++counter)
    // {
    //     printf("%d\t+ %d\t= %d\n", host_numberOne[counter], host_numberTwo[counter], host_addition[counter]);
    // }

    // Clean up, free all device allocated memory.
    free(host_numberOne);
    free(host_numberTwo);
    free(host_addition);
    hipFree(device_numberOne);
    hipFree(device_numberTwo);
    hipFree(device_addition);

    return 0;
}
