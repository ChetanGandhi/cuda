
#include <hip/hip_runtime.h>
// #include <stdio.h>

#define THREADS_COUNT 512

__global__ void add(int *numberOne, int *numberTwo, int *addition)
{
    addition[threadIdx.x] = numberOne[threadIdx.x] + numberTwo[threadIdx.x];
}

void random_ints(int* dest, int count)
{
    int counter = 0;
    for (counter = 0; counter < count; ++counter)
    {
        dest[counter] = rand();
    }
}

int main(void)
{
    // The host variables.
    int *host_numberOne, *host_numberTwo, *host_addition;

    // The device variables.
    int *device_numberOne, *device_numberTwo, *device_addition;

    // Size of variable per block
    int size = THREADS_COUNT * sizeof(int);

    // int counter = 0;

    // Allocate memory on device for device variables.
    hipMalloc((void **)&device_numberOne, size);
    hipMalloc((void **)&device_numberTwo, size);
    hipMalloc((void **)&device_addition, size);

    // Initialize host variables;
    host_numberOne = (int *)malloc(size);
    random_ints(host_numberOne, THREADS_COUNT);

    host_numberTwo = (int *)malloc(size);
    random_ints(host_numberTwo, THREADS_COUNT);

    host_addition = (int *)malloc(size);

    // Copy host variables to device memory.
    hipMemcpy(device_numberOne, host_numberOne, size, hipMemcpyHostToDevice);
    hipMemcpy(device_numberTwo, host_numberTwo, size, hipMemcpyHostToDevice);

    // Invoke add kernel.
    add<<<1, THREADS_COUNT>>>(device_numberOne, device_numberTwo, device_addition);

    // Copy device variable to host memory.
    hipMemcpy(host_addition, device_addition, size, hipMemcpyDeviceToHost);

    // for(counter = 0; counter < THREADS_COUNT; ++counter)
    // {
    //     printf("%d\t+ %d\t= %d\n", host_numberOne[counter], host_numberTwo[counter], host_addition[counter]);
    // }

    // Clean up, free all device allocated memory.
    free(host_numberOne);
    free(host_numberTwo);
    free(host_addition);
    hipFree(device_numberOne);
    hipFree(device_numberTwo);
    hipFree(device_addition);

    return 0;
}
