
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>

__global__ void add(int *numberOne, int *numberTwo, int *addition)
{
    *addition = *numberOne + *numberTwo;
}

int main(void)
{
    // The host variables.
    int host_numberOne, host_numberTwo, host_addition;

    // The device variables.
    int *device_numberOne, *device_numberTwo, *device_addition;

    // Size of variable
    int size = sizeof(int);

    // Allocate memory on device for device variables.
    hipMalloc((void **)&device_numberOne, size);
    hipMalloc((void **)&device_numberTwo, size);
    hipMalloc((void **)&device_addition, size);

    // Initialize host variables;
    host_numberOne = 2;
    host_numberTwo = 7;

    // Copy host variables to device memory.
    hipMemcpy(device_numberOne, &host_numberOne, size, hipMemcpyHostToDevice);
    hipMemcpy(device_numberTwo, &host_numberTwo, size, hipMemcpyHostToDevice);

    // Invoke add kernel.
    add<<<1, 1>>>(device_numberOne, device_numberTwo, device_addition);

    // Copy device variable to host memory.
    hipMemcpy(&host_addition, device_addition, size, hipMemcpyDeviceToHost);

    // Clean up, free all device allocated memory.
    hipFree(device_numberOne);
    hipFree(device_numberTwo);
    hipFree(device_addition);

    return 0;
}
